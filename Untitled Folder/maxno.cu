
    #include <hip/hip_runtime.h>
    #include <stdio.h>
    #include <time.h>

    #define SIZE 5

    __global__ void max(int *a , int *c)
    {
    int i = threadIdx.x;

    *c = a[0];

            if(a[i] > *c)
                    {
                    *c = a[i];
                    }

    }

    int main()
    {
    int i;
    srand(time(NULL));

    int a[SIZE];
    int c;

    int *dev_a, *dev_c;

    hipMalloc((void **) &dev_a, SIZE*sizeof(int));
    hipMalloc((void **) &dev_c, SIZE*sizeof(int));

    for( i = 0 ; i < SIZE ; i++)
    {
    a[i] = rand()% 20 + 1;
    }
    for( i = 0 ; i < SIZE ; i++)
    {
    printf("%d ",a[i]);
    }

    hipMemcpy(dev_a , a, SIZE*sizeof(int),hipMemcpyHostToDevice);
    max<<<1,SIZE>>>(dev_a,dev_c);
    hipMemcpy(&c, dev_c, SIZE*sizeof(int),hipMemcpyDeviceToHost);

    printf(" max =  %d ",c);

   

    return 0;
    }
